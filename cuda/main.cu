#include "hip/hip_runtime.h"
// ******************************************
// implicit time stepping implementation of 2D diffusion problem
// Ben Cumming, CSCS
// C version by Gilles Fourestey, CSCS
// *****************************************

// A small benchmark app that solves the 2D fisher equation using second-order
// finite differences.

// Syntax: ./main nx ny nt t

#include <math.h>
#include <omp.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "check.h"
#include "data.h"
#include "linalg.h"
#include "operators.h"
#include "stats.h"

// ==============================================================================

// read command line arguments
static void readcmdline(struct discretization_t* options, int argc, char* argv[])
{
    if (argc != 5)
    {
        printf("Usage: main nx ny nt t\n");
        printf("  nx  number of gridpoints in x-direction\n");
        printf("  ny  number of gridpoints in y-direction\n");
        printf("  nt  number of timesteps\n");
        printf("  t   total time\n");
        exit(1);
    }

    // read nx
    options->nx = atoi(argv[1]);
    if (options->nx < 1)
    {
        fprintf(stderr, "nx must be positive integer\n");
        exit(-1);
    }

    // read ny
    options->ny = atoi(argv[2]);
    if (options->ny < 1)
    {
        fprintf(stderr, "ny must be positive integer\n");
        exit(-1);
    }

    // read nt
    options->nt = atoi(argv[3]);
    if (options->nt < 1)
    {
        fprintf(stderr, "nt must be positive integer\n");
        exit(-1);
    }
    
    // read total time
    double t = atof(argv[4]);
    if (t < 0)
    {
        fprintf(stderr, "t must be positive real value\n");
        exit(-1);
    }

    // store the parameters
    options->N = options->nx * options->ny;

    // compute timestep size
    options->dt = t / options->nt;
    
    // compute the distance between grid points
    // assume that x dimension has length 1.0
    options->dx = 1./(options->nx - 1);
    
    // set alpha, assume diffusion coefficient D is 1
    options->alpha = (options->dx*options->dx) / (1.*options->dt);
}

// ==============================================================================

namespace gpu
{
	__global__ void main(double* x_new)
	{
		using namespace gpu;

		int nx = options.nx;
		int ny = options.ny;
		int N  = options.N;
		int nt = options.nt;

		// Device malloc has small alignment, so we align manually here.
		double *x_old_u, *bndN_u, *bndS_u, *bndE_u, *bndW_u;
		CUDA_ERR_CHECK(hipMalloc(&x_old_u,  sizeof(double) * nx * ny + (1 << 7)));
		CUDA_ERR_CHECK(hipMalloc(&bndN_u,   sizeof(double) * nx + (1 << 7)));
		CUDA_ERR_CHECK(hipMalloc(&bndS_u,   sizeof(double) * nx + (1 << 7)));
		CUDA_ERR_CHECK(hipMalloc(&bndE_u,   sizeof(double) * ny + (1 << 7)));
		CUDA_ERR_CHECK(hipMalloc(&bndW_u,   sizeof(double) * ny + (1 << 7)));
		x_old = roundPow2(x_old_u, 7);
		bndN = roundPow2(bndN_u, 7);
		bndS = roundPow2(bndS_u, 7);
		bndE = roundPow2(bndE_u, 7);
		bndW = roundPow2(bndW_u, 7);

	    double *b_u, *b;
	    CUDA_ERR_CHECK(hipMalloc(&b_u,      sizeof(double) * N + (1 << 7)));
	    b = roundPow2(b_u, 7);
	    double *deltax_u, *deltax;
	    CUDA_ERR_CHECK(hipMalloc(&deltax_u, sizeof(double) * N + (1 << 7)));
	    deltax = roundPow2(deltax_u, 7);
	    
	    // setting up shmem-cached flops counters
	    flops_diff = 0, flops_blas1 = 0;
	    iters_cg = 0; iters_newton = 0;

		// setting up shmem-cached kernel compute grid configs
		memcpy(ss_sum_kernel::configs, ss_sum_kernel::configs_c, sizeof(config_t) * MAX_CONFIGS);
		memcpy(ss_dot_kernel::configs, ss_dot_kernel::configs_c, sizeof(config_t) * MAX_CONFIGS);
		memcpy(ss_norm2_kernel::configs, ss_norm2_kernel::configs_c, sizeof(config_t) * MAX_CONFIGS);
		ss_fill_kernel::config = ss_fill_kernel::config_c;
		ss_axpy_kernel::config = ss_axpy_kernel::config_c;
		ss_add_scaled_diff_kernel::config = ss_add_scaled_diff_kernel::config_c;
		ss_add_scaled_diff_kernel::config = ss_add_scaled_diff_kernel::config_c;
		ss_scaled_diff_kernel::config = ss_scaled_diff_kernel::config_c;
		ss_scale_kernel::config = ss_scale_kernel::config_c;
		ss_lcomb_kernel::config = ss_lcomb_kernel::config_c;
		ss_copy_kernel::config = ss_copy_kernel::config_c;
		diffusion_interior_grid_points_kernel::config = diffusion_interior_grid_points_kernel::config_c;
		diffusion_east_west_boundary_points_kernel::config = diffusion_east_west_boundary_points_kernel::config_c;
		diffusion_north_south_boundary_points_kernel::config = diffusion_north_south_boundary_points_kernel::config_c;

		// set dirichlet boundary conditions to 0 all around
		ss_fill(x_old,  0, N);
		ss_fill(bndN,   0, nx);
		ss_fill(bndS,   0, nx);
		ss_fill(bndE,   0, ny);
		ss_fill(bndW,   0, ny);
		ss_fill(deltax, 0, N);
	
		// main timeloop
		double tolerance = 1.e-6;
		int timestep;
		for (timestep = 1; timestep <= nt; timestep++)
		{
		    // set x_new and x_old to be the solution
		    ss_copy(x_old, x_new, N);

		    double residual;
		    int    converged = 0;
		    int    it = 1;
		    for ( ; it <= 50; it++)
		    {
		        // compute residual : requires both x_new and x_old
		        diffusion(x_new, b);
		        residual = ss_norm2(b, N);

		        // check for convergence
		        if (residual < tolerance)
		        {
		            converged = 1;
		            break;
		        }

		        // solve linear system to get -deltax
		        bool cg_converged = ss_cg(N, deltax, b, 200, tolerance);

		        // check that the CG solver converged
		        if (!cg_converged) break;

		        // update solution
		        ss_axpy(x_new, -1.0, deltax, N);

		        // print control sum of x_new
		        if (timestep % 50 == 0)
		        {
		        	double sum = ss_sum(x_new, N);
		            printf("sum = %f\n", sum);
				}
		    }
		    iters_newton += it;

		    // output some statistics
		    if (converged && verbose_output)
		        printf("step %d required %d iterations for residual %E\n", timestep, it, residual);
		    if (!converged)
		    {
		        printf("step %d ERROR : nonlinear iterations failed to converge\n", timestep);
		        break;
		    }
		}

		flops_diff_d += flops_diff; flops_blas1_d += flops_blas1;
		iters_cg_d += iters_cg; iters_newton_d += iters_newton;

		free(x_old_u);
		free(bndN_u);
		free(bndS_u);
		free(bndE_u);
		free(bndW_u);
		free(b_u);
		free(deltax_u);
	}
}

int main(int argc, char* argv[])
{
    // read command line arguments
    readcmdline(&cpu::options, argc, argv);
    CUDA_ERR_CHECK(hipMemcpyToSymbol(HIP_SYMBOL(
    	gpu::options), &cpu::options, sizeof(struct discretization_t)));

	int nx = cpu::options.nx;
	int ny = cpu::options.ny;
	int N  = cpu::options.N;
	int nt = cpu::options.nt;

    printf("========================================================================\n");
    printf("                      Welcome to mini-stencil!\n");
    printf("mesh :: %d * %d, dx = %f\n", nx, ny, cpu::options.dx);
    printf("time :: %d, time steps from 0 .. %f\n", nt, nt * cpu::options.dt);
    printf("========================================================================\n");

    // allocate global fields
    double* cpu_x_new  = (double*)malloc(sizeof(double) * nx * ny);
    {
    	using namespace cpu;

		// set the initial condition
		// a circle of concentration 0.1 centred at (xdim/4, ydim/4) with radius
		// no larger than 1/8 of both xdim and ydim
		memset(cpu_x_new, 0, sizeof(double) * nx * ny);
		double xc = 1.0 / 4.0;
		double yc = (ny - 1) * options.dx / 4;
		double radius = fmin(xc, yc) / 2.0;
		for (int j = 0; j < ny; j++)
		{
		    double y = (j - 1) * options.dx;
		    for (int i = 0; i < nx; i++)
		    {
		        double x = (i - 1) * options.dx;
		        if ((x - xc) * (x - xc) + (y - yc) * (y - yc) < radius * radius)
		            cpu_x_new[i + j * nx] = 0.1;
		    }
		}
	}

	CUDA_ERR_CHECK(hipGetDeviceProperties(&cpu::props, 0));
	
	// copy initial solution to GPU
	double* gpu_x_new;
	CUDA_ERR_CHECK(hipMalloc(&gpu_x_new, sizeof(double) * nx * ny));
	CUDA_ERR_CHECK(hipMemcpy(gpu_x_new, cpu_x_new, sizeof(double) * nx * ny, hipMemcpyHostToDevice));
    
    // Calibrating kernels compute grids for the given problem dimensions.
    {
    	determine_optimal_grid_block_config(diffusion_interior_grid_points, 1, nx - 2, ny - 2);
    	determine_optimal_grid_block_config(diffusion_east_west_boundary_points, 1, 1, ny - 2);
		determine_optimal_grid_block_config(diffusion_north_south_boundary_points, 1, nx - 2, 1);
		determine_optimal_grid_block_configs_reduction(ss_sum, 1, N);
		determine_optimal_grid_block_configs_reduction(ss_dot, 1, N);
		determine_optimal_grid_block_configs_reduction(ss_norm2, 1, N);
		determine_optimal_grid_block_config(ss_fill, 2, N, 1);
		determine_optimal_grid_block_config(ss_axpy, 2, N, 1);
		determine_optimal_grid_block_config(ss_add_scaled_diff, 2, N, 1);
		determine_optimal_grid_block_config(ss_scaled_diff, 2, N, 1);
		determine_optimal_grid_block_config(ss_scale, 2, N, 1);
		determine_optimal_grid_block_config(ss_lcomb, 2, N, 1);
		determine_optimal_grid_block_config(ss_copy, 2, N, 1);
	}

	size_t freeGlobalMem, totalGlobalMem;
	CUDA_ERR_CHECK(hipMemGetInfo(&freeGlobalMem, &totalGlobalMem));
	CUDA_ERR_CHECK(hipDeviceSetLimit(hipLimitMallocHeapSize, freeGlobalMem));

    // start timer
    double timespent = -omp_get_wtime();
    
    gpu::main<<<1, 1>>>(gpu_x_new);
    
    CUDA_ERR_CHECK(hipMemcpy(cpu_x_new, gpu_x_new, sizeof(double) * nx * ny, hipMemcpyDeviceToHost));

    // get times
    timespent += omp_get_wtime();
    unsigned long long flops_total = gpu::get_value(gpu::flops_diff_d) + gpu::get_value(gpu::flops_blas1_d);

	using namespace cpu;

    ////////////////////////////////////////////////////////////////////
    // write final solution to BOV file for visualization
    ////////////////////////////////////////////////////////////////////

    // binary data
    {
        FILE* output = fopen("output.bin", "w");
        fwrite(cpu_x_new, sizeof(double), nx * ny, output);
        fclose(output);
    }

    // metadata
    {
        FILE* output = fopen("output.bov", "wb");
        fprintf(output, "TIME: 0.0\n");
        fprintf(output, "DATA_FILE: output.bin\n");
        fprintf(output, "DATA_SIZE: %d, %d, 1\n", nx, ny);
        fprintf(output, "DATA_FORMAT: DOUBLE\n");
        fprintf(output, "VARIABLE: phi\n");
        fprintf(output, "DATA_ENDIAN: LITTLE\n");
        fprintf(output, "CENTERING: nodal\n");
        //fprintf(output, "BYTE_OFFSET: 4\n");
        fprintf(output, "BRICK_SIZE: 1.0 %f 1.0\n", (ny - 1) * options.dx);
        fclose(output);
    }

    // print table sumarizing results
    printf("--------------------------------------------------------------------------------\n");
    printf("simulation took %f seconds (%f GFLOP/s)\n", timespent, flops_total / 1e9 / timespent);
    printf("%u conjugate gradient iterations\n", gpu::get_value(gpu::iters_cg_d));
    printf("%u newton iterations\n", gpu::get_value(gpu::iters_newton_d));
    printf("--------------------------------------------------------------------------------\n");

    // deallocate global fields
    CUDA_ERR_CHECK(hipFree(gpu_x_new));
    free(cpu_x_new);

    printf("Goodbye!\n");

    return 0;
}

