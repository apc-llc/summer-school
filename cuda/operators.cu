#include "hip/hip_runtime.h"
// *****************************************
// operators.f90
// based on min-app code written by Oliver Fuhrer, MeteoSwiss
// modified by Ben Cumming, CSCS
// *****************************************

// Description: Contains simple operators which can be used on 3d-meshes

#include "check.h"
#include "data.h"
#include "operators.h"
#include "stats.h"

#include <thrust/extrema.h>

using namespace thrust::system::cuda::detail;

#define U(j,i)    up[(i) + (j)*nx]
#define S(j,i)    sp[(i) + (j)*nx]
#define X(j,i) x_old[(i) + (j)*nx]

namespace operators
{
	// We redefine dim3 under namespace, because the default one has
	// constructors, which is not allowed for types device variables
	// (dim3 is used as device vars type below to keep kernel compute
	// grid configuration).
	struct dim3
	{
		unsigned int x, y, z;
		
		__device__ operator ::dim3()
		{
			return ::dim3(x, y, z);
		}
	};

	// Use Thrust occupancy calculator to determine the best size of block.
	template<typename T>
	__device__ inline size_t get_optimal_szblock(T kernel)
	{
		using namespace gpu;
	
		struct function_attributes_t attrs;
		{
		    hipFuncAttributes funcAttrs;
		    CUDA_ERR_CHECK(hipFuncGetAttributes(&funcAttrs, reinterpret_cast<const void*>(kernel)));
		    attrs.constSizeBytes = funcAttrs.constSizeBytes;
		    attrs.localSizeBytes = funcAttrs.localSizeBytes;
		    attrs.maxThreadsPerBlock = funcAttrs.maxThreadsPerBlock;
		    attrs.numRegs = funcAttrs.numRegs;
		    attrs.sharedSizeBytes = funcAttrs.sharedSizeBytes;
		}
		struct device_properties_t props;
		{
		    props.major = gpuProps.major;
		    memcpy(&props.maxGridSize, &gpuProps.maxGridSize, sizeof(int) * 3);
		    props.maxThreadsPerBlock = gpuProps.maxThreadsPerBlock;
		    props.maxThreadsPerMultiProcessor = gpuProps.maxThreadsPerMultiProcessor;
		    props.minor = gpuProps.minor;
		    props.multiProcessorCount = gpuProps.multiProcessorCount;
		    props.regsPerBlock = gpuProps.regsPerBlock;
		    props.sharedMemPerBlock = gpuProps.sharedMemPerBlock;
		    props.warpSize = gpuProps.warpSize;
		}
		return block_size_with_maximum_potential_occupancy(attrs, props);
	}

	template<typename T>
	__device__ inline void get_optimal_grid_block_config(T kernel,
		int nx, int ny, dim3& grid, dim3& blocks)
	{
		size_t szblock = get_optimal_szblock(kernel);

		grid.x = 1; grid.y = 1; grid.z = 1;
		blocks.x = 1; blocks.y = 1; blocks.z = 1;

		if (szblock > nx)
		{
		    blocks.x = nx;
		    blocks.y = min(ny, (int)szblock / blocks.x);
		    grid.y = ny / blocks.y;
		    if (ny % blocks.y) grid.y++;
		}
		else
		{
		    blocks.x = szblock;
		    grid.x = nx / blocks.x;
		    if (nx % blocks.x) grid.x++;
		    grid.y = ny;
		}
	}

	namespace diffusion_interior_grid_points
	{
		__global__ void kernel(const double* up, double* sp)
		{
			using namespace gpu;

			double dxs   = 1000. * options.dx * options.dx;
			double alpha = options.alpha;
			int    iend  = options.nx - 1;
			int    jend  = options.ny - 1;

			int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
			int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

			if ((i >= iend) || (j >= jend)) return;

			int nx = options.nx;

			S(j, i) = -(4. + alpha) * U(j,i)              // central point
						            + U(j,i-1) + U(j,i+1) // east and west
						            + U(j-1,i) + U(j+1,i) // north and south

						            + alpha * X(j,i)
						            + dxs * U(j,i) * (1.0 - U(j,i));
		}

		__device__ dim3 grid, blocks;
		__device__ bool grid_block_init = false;
	}

	namespace diffusion_east_west_boundary_points
	{
		__global__ void kernel(const double* up, double* sp)
		{
			using namespace gpu;

			double dxs   = 1000. * options.dx * options.dx;
			double alpha = options.alpha;
			int    jend  = options.ny - 1;

			int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

			if (j >= jend) return;

			int nx = options.nx;

			int i = options.nx - 1;

			S(j, i) = -(4. + alpha) * U(j,i)
					                + U(j, i - 1) + U(j - 1, i) + U(j + 1, i)

					                + alpha * X(j, i) + bndE[j]
					                + dxs * U(j, i) * (1.0 - U(j, i));

			i = 0;

			S(j, i) = -(4. + alpha) * U(j, i)
					                + U(j, i + 1) + U(j - 1, i) + U(j + 1, i)

					                + alpha * X(j, i) + bndW[j]
					                + dxs * U(j, i) * (1.0 - U(j, i));
		}

		__device__ dim3 grid, blocks;
		__device__ bool grid_block_init = false;
	}

	namespace diffusion_north_south_boundary_points
	{
		__global__ void kernel(const double* up, double* sp)
		{
			using namespace gpu;

			double dxs   = 1000. * options.dx * options.dx;
			double alpha = options.alpha;
			int    iend  = options.nx - 1;

			int i = blockIdx.x * blockDim.x + threadIdx.x + 1;

			if (i >= iend) return;

			int nx = options.nx;

			int j = options.ny - 1;

			S(j, i) = -(4. + alpha) * U(j, i)
					                + U(j, i - 1) + U(j, i + 1) + U(j - 1, i)

					                + alpha * X(j, i) + bndN[i]
					                + dxs * U(j, i) * (1.0 - U(j, i));

			j = 0;

			S(j, i) = -(4. + alpha) * U(j, i)
					                + U(j, i - 1) + U(j, i + 1) + U(j + 1, i)

					                + alpha * X(j, i) + bndS[i]
					                + dxs * U(j, i) * (1.0 - U(j, i));
		}

		__device__ dim3 grid, blocks;
		__device__ bool grid_block_init = false;
	}

	namespace diffusion_corner_points
	{
		__global__ void kernel(const double* up, double* sp)
		{
			using namespace gpu;

			double dxs   = 1000. * options.dx * options.dx;
			double alpha = options.alpha;

			int nx = options.nx;

			{
				int j = options.ny - 1;
				{
					int i = 0; // NW corner
					S(j, i) = -(4. + alpha) * U(j, i)
					            + U(j, i + 1) + U(j - 1, i)

					            + alpha * X(j, i) + bndW[j] + bndN[i]
					            + dxs * U(j, i) * (1.0 - U(j, i));
				}
				{
					int i = options.nx - 1; // NE corner
					S(j, i) = -(4. + alpha) * U(j, i)
					            + U(j, i - 1) + U(j - 1, i)
					            + alpha * X(j, i) + bndE[j] + bndN[i]
					            + dxs * U(j, i) * (1.0 - U(j, i));
				}
			}
			{
				int j = 0;
				{
					int i = 0; // SW corner
					S(j, i) = -(4. + alpha) * U(j, i)
					            + U(j, i + 1) + U(j + 1, i)
					            + alpha * X(j, i) + bndW[j] + bndS[i]
					            + dxs * U(j, i) * (1.0 - U(j, i));
				}
				{
					int i = options.nx - 1; // SE corner
					S(j, i) = -(4. + alpha) * U(j, i)
					            + U(j, i - 1) + U(j + 1, i)
					            + alpha * X(j, i) + bndE[j] + bndS[i]
					            + dxs * U(j, i) * (1.0 - U(j, i));
				}
			}
		}
	}

	__device__ void diffusion(const double* up, double* sp)
	{
		using namespace gpu;

		int nx = options.nx;
		int ny = options.ny;

		// Launch kernel for parallel processing of interior points.
		{
			using namespace diffusion_interior_grid_points;
			if (!grid_block_init)
			{
				get_optimal_grid_block_config(kernel, nx - 2, ny - 2, grid, blocks);
				grid_block_init = true;
			}
			CUDA_LAUNCH_ERR_CHECK(kernel<<<grid, blocks>>>(up, sp));
		}
	
		// Launch kernels for parallel processing of boundary points.
		{
			using namespace diffusion_east_west_boundary_points;
			if (!grid_block_init)
			{
				get_optimal_grid_block_config(kernel, 1, ny - 2, grid, blocks);
				grid_block_init = true;
			}
			CUDA_LAUNCH_ERR_CHECK(kernel<<<grid, blocks>>>(up, sp));
		}
		{
			using namespace diffusion_north_south_boundary_points;
			if (!grid_block_init)
			{
				get_optimal_grid_block_config(kernel, nx - 2, 1, grid, blocks);
				grid_block_init = true;
			}
			CUDA_LAUNCH_ERR_CHECK(kernel<<<grid, blocks>>>(up, sp));
		}
	
		// Launch kernel for single-threaded processing of corner points.
		CUDA_LAUNCH_ERR_CHECK(diffusion_corner_points::kernel<<<1, 1>>>(up, sp));
	
		// Accumulate the flop counts
		// 8 ops total per point
		flops_diff +=
			+ 12 * (options.nx - 2) * (options.ny - 2) // interior points
			+ 11 * (options.nx - 2  +  options.ny - 2) // NESW boundary points
			+ 11 * 4;                                  // corner points}
	}
}

__global__ void diffusion(const double* up, double* sp) { operators::diffusion(up, sp); }

